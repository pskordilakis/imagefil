#include "hip/hip_runtime.h"
#include "filter.h"

#include <hip/hip_runtime.h>

texture<unsigned char, 2> dataIn;
texture<unsigned char, 2> dataOut;

__constant__ int kernel_3x3[9];
__constant__ int kernel_sum[1];
__constant__ int total_planes[1];
__constant__ int current_plane[1];

__global__ void filter_3x3_kernel(unsigned char *data, bool dstOut) {
	
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * gridDim.x * blockDim.x;
	
	int ul, u, ur, l, c, r, dl, d, dr;
	if(dstOut) {
		ul = tex2D(dataIn, x - 1, y - 1) * (kernel_3x3[0]);
		u = tex2D(dataIn, x, y - 1) * (kernel_3x3[1]);
		ur = tex2D(dataIn, x + 1, y - 1) * (kernel_3x3[2]);
		l = tex2D(dataIn, x - 1, y) * (kernel_3x3[3]);
		c = tex2D(dataIn, x, y) * (kernel_3x3[4]);
		r = tex2D(dataIn, x + 1, y) * (kernel_3x3[5]);
		dl = tex2D(dataIn, x - 1, y + 1) * (kernel_3x3[6]);
		d = tex2D(dataIn, x, y + 1) * (kernel_3x3[7]);
		dr = tex2D(dataIn, x + 1, y + 1) * (kernel_3x3[8]);
	} else {
		ul = tex2D(dataOut, x - 1, y - 1) * (kernel_3x3[0]);
		u = tex2D(dataOut, x, y - 1) * (kernel_3x3[1]);
		ur = tex2D(dataOut, x + 1, y - 1) * (kernel_3x3[2]);
		l = tex2D(dataOut, x - 1, y) * (kernel_3x3[3]);
		c = tex2D(dataOut, x, y) * (kernel_3x3[4]);
		r = tex2D(dataOut, x + 1, y) * (kernel_3x3[5]);
		dl = tex2D(dataOut, x - 1, y + 1) * (kernel_3x3[6]);
		d = tex2D(dataOut, x, y + 1) * (kernel_3x3[7]);
		dr = tex2D(dataOut, x + 1, y + 1) * (kernel_3x3[8]);
	}
	
	data[offset] = (unsigned char)((ul + u + ur + l + c + r + dl + d + dr)/(kernel_sum[0]));
}

__global__ void deinterleave_planes(unsigned char *original, unsigned char *data) {
	
	//each thread split one pixel
	
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int out_offset = x + y * gridDim.x * blockDim.x;
	int in_offset = x * total_planes[0] +  current_plane[0] + y * gridDim.x * blockDim.x * total_planes[0];
	
	data[out_offset] = original[in_offset]; 
}

__global__ void interleave_planes(unsigned char *data, unsigned char *out) {
	
	//each thread interleave one pixel
	
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int in_offset = x + y * gridDim.x * blockDim.x;
	int out_offset = x * total_planes[0] +  current_plane[0] + y * gridDim.x * blockDim.x * total_planes[0];
	
	out[out_offset] = data[in_offset];
}


extern "C" void apply_filter_rgb(image_t *in, filter_t *f, image_t *out, int ntimes) {
	
	int plane, planes, time;
	bool dstOut;
	
	planes = 3; 
	
	dim3 f_numBlocks(in->width*planes/30, in->height/30);
	dim3 numBlocks(in->width/30, in->height/30);
	dim3 threadsPerBlock(30, 30);

	unsigned char *original_src, *data1, *data2, *final_out;
	
	if(hipMalloc((void **) &original_src, in->height*in->width*planes) != hipSuccess) {
		perror("Could not allocate original_src");
		return;
	}
	
	if(hipMalloc((void **) &data1, in->height*in->width) != hipSuccess) {
		perror("Could not allocate data1");
		hipFree(original_src);
		return;
	}
	
	if(hipMalloc((void **) &data2, in->height*in->width) != hipSuccess) {
		perror("Could not allocate data2");
		hipFree(original_src);
		hipFree(data1);
		return;
	}
	
	if(hipMalloc((void **) &final_out, in->height*in->width*planes) != hipSuccess) {
		perror("Could not allocate final_out");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		return;
	}
	
	if(hipMemcpy(original_src, in->data[0], in->height*in->width*planes*sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess) {
		perror("Could copy image to device");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		hipFree(final_out);
		return;
	}
	
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	
	if(hipBindTexture2D( NULL, dataIn, data1, desc, in->width, in->height, in->width*sizeof(unsigned char)) != hipSuccess) {
		perror("Could not bind texture dataIn");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		hipFree(final_out);
		return;
	}

	if(hipBindTexture2D( NULL, dataOut, data2, desc, in->width, in->height, in->width*sizeof(unsigned char)) != hipSuccess) {
		perror("Could not bind texture dataOut");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		hipFree(final_out);
		return;
	}
	
	if(hipMemcpyToSymbol(HIP_SYMBOL(kernel_3x3), f->kernel[0], f->rows*f->cols*sizeof(int), 0, hipMemcpyHostToDevice ) != hipSuccess) {
		perror("Could not copy kernel to constant");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		hipFree(final_out);
		return;
	}
	
	if(hipMemcpyToSymbol(HIP_SYMBOL(kernel_sum), &(f->sum), sizeof(int), 0, hipMemcpyHostToDevice ) != hipSuccess) {
		perror("Could not copy kernel sum to constant");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		hipFree(final_out);
		return;
	}
	
	if(hipMemcpyToSymbol(HIP_SYMBOL(total_planes), &(planes), sizeof(int), 0, hipMemcpyHostToDevice ) != hipSuccess) {
		perror("Could not copy image height to constant");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		hipFree(final_out);
		return;
	}
	
	for(plane = 0; plane < planes; plane++) {
		
		dstOut = true;
		
		if(hipMemcpyToSymbol(HIP_SYMBOL(current_plane), &(plane), sizeof(int), 0, hipMemcpyHostToDevice ) != hipSuccess) {
			perror("Could not copy image height to constant");
			hipFree(original_src);
			hipFree(data1);
			hipFree(data2);
			hipFree(final_out);
			return;
		}
		
		//deinterleave data
		deinterleave_planes<<<numBlocks, threadsPerBlock>>>(original_src, data1);
		
		if(hipSuccess != hipGetLastError()) {
			perror("deinteleave kernel error");
			break;
		}
		
		//filter
		for(time=0; time < ntimes; time++) {
		
			if(dstOut) {
				filter_3x3_kernel<<<numBlocks, threadsPerBlock>>>(data2, dstOut);
			} else { 
				filter_3x3_kernel<<<numBlocks, threadsPerBlock>>>(data1, dstOut);
			}
			
			if(hipSuccess != hipGetLastError()) {
				perror("filter kernel error");
				break;
			}
			
			dstOut = !dstOut;
		}
		
		//interleave out
		if(dstOut) {
			interleave_planes<<<numBlocks, threadsPerBlock>>>(data1, final_out);
		} else {
			interleave_planes<<<numBlocks, threadsPerBlock>>>(data2, final_out);
		}
		
		
		
		if(hipSuccess != hipGetLastError()) {
			perror("inteleave kernel error");
			break;
		}
	}
	
	if(hipMemcpy(out->data[0], final_out, out->height*out->width*planes*sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess) {
		perror("hipMemcpy error");
		hipFree(original_src);
		hipFree(data1);
		hipFree(data2);
		hipFree(final_out);
		return;
	}
	
	hipUnbindTexture( dataIn );
	hipUnbindTexture( dataOut );
	
	hipFree(original_src);
	hipFree(data1);
	hipFree(data2);
	hipFree(final_out);
	
}	

extern "C" void apply_filter_gs(image_t *in, filter_t *f, image_t *out, int ntimes) {
	
	int time;
	bool dstOut = true; 
	
	dim3 numBlocks(in->width/30, in->height/30);
	dim3 threadsPerBlock(30, 30);
	
	unsigned char *data1, *data2;
	
	if(hipMalloc((void **) &data1, in->height*in->width) != hipSuccess) {
		perror("Could not allocate data1");
		return;
	}
	
	if(hipMalloc((void **) &data2, in->height*in->width) != hipSuccess) {
		perror("Could not allocate data2");
		hipFree(data1);
		return;
	}
	
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	
	if(hipBindTexture2D( NULL, dataIn, data1, desc, in->width, in->height, in->width*sizeof(unsigned char)) != hipSuccess) {
		perror("Could not bind texture dataIn");
		hipFree(data1);
		hipFree(data2);
		return;
	}
	
	if(hipMemcpy(data1, in->data[0], in->height*in->width*sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess) {
		perror("Could copy image to device");
		hipFree(data1);
		hipFree(data2);
		return;
	}
	
	if(hipBindTexture2D( NULL, dataOut, data2, desc, in->width, in->height, in->width*sizeof(unsigned char)) != hipSuccess) {
		perror("Could not bind texture dataOut");
		hipFree(data1);
		hipFree(data2);
		return;
	}
	
	if(hipMemcpyToSymbol(HIP_SYMBOL(kernel_3x3), f->kernel[0], f->rows*f->cols*sizeof(**f->kernel), 0, hipMemcpyHostToDevice ) != hipSuccess) {
		perror("Could not copy kernel to constant");
		hipFree(data1);
		hipFree(data2);
		return;
	}
	
	int sum = f->sum;
	if(hipMemcpyToSymbol(HIP_SYMBOL(kernel_sum), &(sum), sizeof(int), 0, hipMemcpyHostToDevice ) != hipSuccess) {
		perror("Could not copy kernel sum to constant");
		hipFree(data1);
		hipFree(data2);
		return;
	}
	
	//filter
	for(time=0; time < ntimes; time++) {
	
		if(dstOut) {
			filter_3x3_kernel<<<numBlocks, threadsPerBlock>>>(data2, dstOut);
		} else { 
			filter_3x3_kernel<<<numBlocks, threadsPerBlock>>>(data1, dstOut);
		}
		
		if(hipSuccess != hipGetLastError()) {
			perror("filter kernel error");
			break;
		}
		
		dstOut = !dstOut;
	}
	
	if(dstOut) {
		if(hipMemcpy(out->data[0], data1, out->height*out->width*sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess) {
			perror("hipMemcpy error");
			hipFree(data1);
			hipFree(data2);
			return;
		}	
	} else {
		if(hipMemcpy(out->data[0], data2, out->height*out->width*sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess) {
			perror("hipMemcpy error");
			hipFree(data1);
			hipFree(data2);
			return;
		}
	}

	hipUnbindTexture( dataIn );
	hipUnbindTexture( dataOut );
	
	hipFree(data1);
	hipFree(data2);
}		

